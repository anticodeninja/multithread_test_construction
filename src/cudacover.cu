#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cudacover.h"

#define THREAD_PER_BLOCK 256

__global__
void kernel(int cover_koef, int features, char *lset, int lset_len, char *block, int block_len, int *results, int *results_counter)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if (index < block_len) {
        bool coverAll = true;

        for (int i = 0; i < lset_len; ++i) {
            int cover = 0;

            for (int j = 0; j < features; ++j) {
                if (lset[features * i + j] && block[features * index + j]) {
                    cover += 1;
                }
            }

            if (cover < cover_koef) {
                coverAll = false;
                break;
            }
        }

        if (coverAll) {
            results[atomicAdd(results_counter, 1)] = index;
        }
    }
}

cudacover_result_t cudacover_init(cudacover_t** ctx,
                                  int lset_len,
                                  int features,
                                  int block_len) {
    cudacover_t* temp = (cudacover_t*)malloc(sizeof(cudacover_t));
    if (temp == NULL) {
        return CUDA_COVER_RESULT_ERR;
    }

    temp->cover_koef = 1;
    temp->lset_len = lset_len;
    temp->features = features;
    temp->block_len = block_len;
    temp->lset = (char*)malloc(lset_len * features * sizeof(char));
    temp->block = (char*)malloc(block_len * features * sizeof(char));
    temp->results = (int*)malloc(block_len * sizeof(int));
    temp->results_counter = 0;

    hipMalloc(&temp->__lset, lset_len * features * sizeof(char));
    hipMalloc(&temp->__block, block_len * features * sizeof(char));
    hipMalloc(&temp->__results, block_len * sizeof(int));
    hipMalloc(&temp->__results_counter, sizeof(int));
    temp->__lset_uploaded = false;

    *ctx = temp;
    return CUDA_COVER_RESULT_OK;
}

cudacover_result_t cudacover_check(cudacover_t* ctx,
                                   int block_len) {
    if (ctx == NULL) {
        return CUDA_COVER_RESULT_ERR;
    }

    if (!ctx->__lset_uploaded) {
        ctx->__lset_uploaded = true;
        hipMemcpy(ctx->__lset, ctx->lset, ctx->lset_len * ctx->features * sizeof(char), hipMemcpyHostToDevice);
    }
    hipMemcpy(ctx->__block, ctx->block, block_len * ctx->features * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(ctx->__results_counter, &ctx->results_counter, sizeof(int), hipMemcpyHostToDevice);

    dim3 blocks = dim3((block_len + THREAD_PER_BLOCK - 1)/THREAD_PER_BLOCK);
    dim3 threads = dim3(THREAD_PER_BLOCK);
    kernel<<<blocks, threads>>>(ctx->cover_koef,
                                ctx->features,
                                ctx->__lset,
                                ctx->lset_len,
                                ctx->__block,
                                block_len,
                                ctx->__results,
                                ctx->__results_counter);

    hipMemcpy(&ctx->results_counter, ctx->__results_counter, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(ctx->results, ctx->__results, ctx->results_counter * sizeof(int), hipMemcpyDeviceToHost);

    return CUDA_COVER_RESULT_OK;
}

cudacover_result_t cudacover_free(cudacover_t** ctx) {
    cudacover_t* temp = *ctx;
    if (temp == NULL) {
        return CUDA_COVER_RESULT_ERR;
    }

    hipFree(temp->__lset);
    hipFree(temp->__block);
    hipFree(temp->__results_counter);
    hipFree(temp->__results);

    free(temp->lset);
    free(temp->block);
    free(temp->results);
    free(temp);

    *ctx = NULL;
    return CUDA_COVER_RESULT_OK;
}
